
#include <hip/hip_runtime.h>
#define LN2_INV 1.4426950408889634
#define TWO_PI 6.283185307179586
#define PI 3.141592653589793
#define E 2.718281828459045


__forceinline__ double __device__ exponent(double x)
{
    return exp(x);
}

__forceinline__ float __device__ exponent(float x)
{
    return expf(x);
}

__forceinline__ double __device__ cosine(double x)
{
    return cos(x);
}

__forceinline__ float __device__ cosine(float x)
{
    return cosf(x);
}

__forceinline__ double __device__ sine(double x)
{
    return sin(x);
}

__forceinline__ float __device__ sine(float x)
{
    return sinf(x);
}

template<class T>
__forceinline__ T __device__ clamp(const T& x, const T& lower, const T& upper) 
{
    return min(upper, max(x, lower));
}

template <typename T>
void __device__  render_fractal(unsigned char* nv12_yuv, unsigned int height, unsigned int width, unsigned int stride,
                               T x_0, T x_step, T y_1, T y_step, T cutoff_value_sq, unsigned int max_iterations, 
                               unsigned int x, unsigned int y)
{
    if(x>=width || y>= height)
        return;
        
    unsigned i;

    T initial_real = x_0 + x*x_step;
    T initial_imag = y_1 - y*y_step;
    T current_real = initial_real;
    T current_imag = initial_imag;
    
    T current_abs_sq = current_imag*current_imag+current_real*current_real;
    
    for(i=0; i<max_iterations &&  current_abs_sq < cutoff_value_sq;++i)
    {
        T current_exp_arg = exponent(-current_real);
        T current_square_imag = 2 * current_real * current_imag;
        T cos_current_imag = cosine(current_imag);
        T sin_current_imag = sine(current_imag);
        current_real = current_real*current_real - current_imag*current_imag + current_exp_arg*(cos_current_imag*initial_real + sin_current_imag*initial_imag);
        current_imag = current_exp_arg*(cos_current_imag*initial_imag - sin_current_imag*initial_real) + current_square_imag;
        current_abs_sq = current_imag*current_imag+current_real*current_real;
    }
    
    float luma,cr,cb;
    /* ---- Colorization -----*/
    if(i<max_iterations)
    {
        double ln_cutoff_val_sq = logf(cutoff_value_sq);
        double smoothing_factor = LN2_INV*(log(0.5*log(current_abs_sq)-0.5*log(ln_cutoff_val_sq)));
        double smoothed_iterations = i+1-smoothing_factor;
        // double variable = log(smoothed_iterations*((E-1)/(max_iterations+1-smoothing_factor))+1)*TWO_PI;
        double variable = smoothed_iterations/max_iterations*PI;
        
        luma = 50 + (unsigned char)clamp((200.0 * variable),0.0,200.0);
        cr = (unsigned char)clamp((235.5*cosine(0.5*variable)),16.0,235.0);
        cb = (unsigned char)clamp((235.5*sine(3.0*variable)),16.0,128.0);
    }
    else
    {
        luma = 16;
        cr = 128;
        cb = 128;
    }

    nv12_yuv[y*stride+x] = luma;
    
    if(((x & 1u) == 0u) && ((y & 1u) == 0u))
    {        
        nv12_yuv[height*stride+y*stride/2+x] = cr;
        nv12_yuv[height*stride+y*stride/2+x+1]= cb;
    }
    
}

extern "C" __global__ void render_fractal_float(unsigned char* nv12_yuv,unsigned int height, unsigned int width, unsigned int stride,
                                                float x_0, float x_step, float y_1, float y_step,float cutoff_value_sq,
                                                unsigned int max_iterations)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    
    render_fractal(nv12_yuv,height,width,stride,x_0,x_step,y_1,y_step,cutoff_value_sq,max_iterations,x,y);
}

extern "C" __global__ void render_fractal_double(unsigned char* nv12_yuv, unsigned int height, unsigned int width, unsigned int stride,
                                                 double x_0, double x_step, double y_1, double y_step, double cutoff_value_sq,
                                                 unsigned int max_iterations)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    
    render_fractal(nv12_yuv,height,width,stride,x_0,x_step,y_1,y_step,cutoff_value_sq,max_iterations,x,y);
}

